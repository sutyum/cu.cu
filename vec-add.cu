
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// Each thead performs one pa-r-wose addition
__global__ 
void vecAddKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

__global__ 
void vecRangeKernel(float* A, float start, int n, float inc) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {
    A[i] = start + inc * i;
  }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n) { // host pointers
  int size = n * sizeof(float);
  float *A_d, *B_d, *C_d; // device pointers

  // Step 1: Allocate device memory for A, B, and C
  // Copy A and B to device memory
  hipMalloc((void**) &A_d, size);
  hipMalloc((void**) &B_d, size);
  hipMalloc((void**) &C_d, size);

  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  // Step 2: Call kernel - to launch a grid of threads
  // to perform the actual vector addition
  vecRangeKernel<<<ceil(n/256.0), 256>>>(A_d, 0, n, 2);
  vecRangeKernel<<<ceil(n/256.0), 256>>>(B_d, 1, n, 1);

  vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);
  
  // Step 3: Copy C from the device memory
  // Free device vectors
  hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main() {
  #define VEC_SIZE 100000

  float* A = (float*) malloc(sizeof(float) * VEC_SIZE);
  float* B = (float*) malloc(sizeof(float) * VEC_SIZE);
  float* C = (float*) malloc(sizeof(float) * VEC_SIZE);

  const int print_n = 3;
  vecAdd(A, B, C, VEC_SIZE);
  for (int i = 0; i < print_n; ++i) {
    printf("C[%d]=%.0f\n", i+1, C[i]);
  }
  printf("...\n");
  for (int i = VEC_SIZE - print_n; i < VEC_SIZE; ++i) {
    printf("C[%d]=%.0f\n", i+1, C[i]);
  }

  free(A);
  free(B);
  free(C);

  printf("Done!\n");

  return 0;
}